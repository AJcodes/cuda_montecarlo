#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <iomanip>
#include <iostream>

#define BLOCKSIZE 64

hipError_t mcCuda(double *val, double *val1, const int& num_sims, const double& S, const double& K, const double& r, const double& v, const double& T);

__global__ void init(unsigned int seed, hiprandState_t* states, double *normal, const int num_sims) {
	int id = blockIdx.x * BLOCKSIZE + threadIdx.x;
	if (id < num_sims) {
		hiprand_init(seed, id, 0, &states[id]);
		normal[id] = hiprand_normal_double(&states[id]);
	}
}

__global__ void mcKernel(double *normal, const int num_sims, const double S, const double K, const double r, const double v, const double T, double *val, double *val1) {
	__shared__ double c[BLOCKSIZE];
	__shared__ double p[BLOCKSIZE];
    double S_adjust = S * exp(T*(r-0.5*v*v));
	double S_cur = 0.0;
	double payoff_sum = 0.0;
	double payoff_sum1 = 0.0;
	double call_temp = 0.0;
	double put_temp = 0.0;
	int id = blockIdx.x * BLOCKSIZE + threadIdx.x;
	int bid = blockIdx.x;

	if (id < num_sims) {
			double gauss_bm = normal[id];
			S_cur = S_adjust * exp(sqrt(v*v*T)*gauss_bm);
			payoff_sum = max(S_cur - K, 0.0);
			payoff_sum1 = max(K - S_cur, 0.0);
			c[threadIdx.x] = (payoff_sum / num_sims) * exp(-r*T);
			p[threadIdx.x] = (payoff_sum1 / num_sims) * exp(-r*T);
			__syncthreads();

			for (int i = 0; i <BLOCKSIZE; ++i) {
				call_temp += c[i];
				put_temp += p[i];
			}
	}

	val[bid] = call_temp;
	val1[bid] = put_temp;
}

int main() {
	int num_sims = 100000;   // Number of simulated asset paths                                                       
	double S = 100.0;  // Option price                                                                                  
	double K = 100.0;  // Strike price                                                                                  
	double r = 0.05;   // Risk-free rate (5%)                                                                           
	double v = 0.2;    // Volatility of the underlying (20%)                                                            
	double T = 1.0;    // One year until expiry
	double *val = 0;
	double *val1 = 0;

	val = (double *) malloc(num_sims * sizeof(double));
	val1 = (double *) malloc(num_sims * sizeof(double));

    hipError_t cudaStatus = mcCuda(val, val1, num_sims, S, K, r, v, T);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t mcCuda(double *val, double *val1, const int& num_sims, const double& S, const double& K, const double& r, const double& v, const double& T) {
	double * dev_val = 0;
	double * dev_val1 = 0;
	double * dev_normal = 0;
    hipError_t cudaStatus;
	hiprandState_t* states;
	dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid(ceil(float(num_sims) / float(BLOCKSIZE)));
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEvent_t start1, stop1;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	float milliseconds = 0.0f;
	float milliseconds1 = 0.0f;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**) &states, num_sims * sizeof(hiprandState_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_val, num_sims * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_val1, num_sims * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_normal, num_sims * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	init<<<dimGrid, dimBlock>>>(time(0), states, dev_normal, num_sims);

	hipEventRecord(start);
    mcKernel<<<dimGrid, dimBlock>>>(dev_normal, num_sims, S, K, r, v, T, dev_val, dev_val1);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "mcKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching mcKernel!\n", cudaStatus);
        goto Error;
    }

	cudaStatus = hipMemcpy(val, dev_val, num_sims * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(val1, dev_val1, num_sims * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	hipEventRecord(start1);
	double call = 0;
	for (int i = 0; i < ceil(float(num_sims) / float(BLOCKSIZE)); i++) {
		call += val[i];
	}
	double put = 0;
	for (int i = 0; i < ceil(float(num_sims) / float(BLOCKSIZE)); i++) {
		put += val1[i];
	}

	hipEventRecord(stop1);
	hipEventSynchronize(stop1);

	hipEventElapsedTime(&milliseconds, start, stop);
	hipEventElapsedTime(&milliseconds1, start1, stop1);
	std::cout << "Number of Paths: " << num_sims << std::endl;
	std::cout << "Underlying:      " << S << std::endl;
	std::cout << "Strike:          " << K << std::endl;
	std::cout << "Risk-Free Rate:  " << r << std::endl;
	std::cout << "Volatility:      " << v << std::endl;
	std::cout << "Maturity:        " << T << std::endl;
	std::cout << "================================" << std::endl;
	std::cout << "Call Price:	   " << call << std::endl;
	std::cout << "Put Price:	   " << put << std::endl;
	std::cout << "================================" << std::endl;
	std::cout << "Execution Time : " << (milliseconds + milliseconds1) / 1000 << " seconds" << std::endl;
	std::cout << "Execution Time (GPU only): " << (milliseconds) / 1000 << " seconds" << std::endl;
	std::cout << "Execution Time (Summation only): " << (milliseconds1) / 1000 << " seconds" << std::endl;
	std::cout << "Effective Bandwidth : " << (num_sims*sizeof(double)*2) / ((milliseconds + milliseconds1) / 1000) << " GB/s" << std::endl;

Error:
    hipFree(dev_val);
	hipFree(dev_val1);
	hipFree(dev_normal);
	hipFree(states);
    
    return cudaStatus;
}
